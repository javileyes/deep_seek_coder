#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    hipblasHandle_t handle;
    hipblasStatus_t status;

    // Inicializar CUBLAS
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS no está instalado o no funciona correctamente.\n");
        return -1;
    }

    // Ejemplo sencillo: Escalar un vector
    const int n = 10;
    float alpha = 2.0f;
    float x[n] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0};
    float *d_x;

    // Asignar memoria en la GPU
    hipMalloc((void **)&d_x, n * sizeof(float));

    // Copiar datos al dispositivo
    hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);

    // Escalar el vector x por alpha y almacenar el resultado en x
    hipblasSscal(handle, n, &alpha, d_x, 1);

    // Copiar los resultados de vuelta a la memoria del host
    hipblasGetVector(n, sizeof(float), d_x, 1, x, 1);

    // Limpiar
    hipFree(d_x);
    hipblasDestroy(handle);

    printf("CUBLAS está instalado y funcionando correctamente.\n");
    return 0;
}
