#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount > 0) {
        printf("GPU is available\n");
    } else {
        printf("GPU is not available\n");
    }
    return 0;
}

